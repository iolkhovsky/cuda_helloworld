#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2048 * 2048 

__global__ void saxpy(int * a, int * b, int * c)
{
    int stride = blockDim.x * gridDim.x;
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += stride)
        c[i] = 2 * a[i] + b[i];
}

__global__ void init(int * a, int * b, int * c)
{
    int stride = blockDim.x * gridDim.x;
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += stride) {
        a[i] = 2;
        b[i] = 1;
        c[i] = 0;
    }
}

int main()
{
    int deviceId;
    hipGetDevice(&deviceId);

    int threads_per_block = 128;
    int number_of_blocks = (N / threads_per_block) + 1;

    float *a, *b, *c;

    int size = N * sizeof (int);

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    hipMemPrefetchAsync(a, size, deviceId);
    hipMemPrefetchAsync(b, size, deviceId);
    hipMemPrefetchAsync(c, size, deviceId);

    init<<<number_of_blocks, threads_per_block>>>(a, b, c);
    saxpy <<< number_of_blocks, threads_per_block >>> ( a, b, c );

    hipMemPrefetchAsync(c, size, hipCpuDeviceId);
    
    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");

    hipFree( a ); hipFree( b ); hipFree( c );
}
