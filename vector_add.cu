
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

hipDeviceProp_t getProps() {
  int deviceId;
  hipGetDevice(&deviceId);
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);
  return props;
}

int main()
{
  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  size_t threadsPerBlock;
  size_t numberOfBlocks;
   
  threadsPerBlock = 1;
  numberOfBlocks = 1;
  hipDeviceProp_t gpuProps = getProps();
  int threads_per_sm = N / gpuProps.multiProcessorCount;
  if (N % gpuProps.multiProcessorCount)
      threads_per_sm++;
      
  if (threads_per_sm > 1024) {
      int k = threads_per_sm / 1024;
      if (threads_per_sm % 1024)
          k++;
      numberOfBlocks = k * gpuProps.multiProcessorCount;
      threadsPerBlock = N / numberOfBlocks;
      if (N % numberOfBlocks)
          threadsPerBlock++;
  } else {
      threadsPerBlock = threads_per_sm;
      numberOfBlocks = gpuProps.multiProcessorCount;
  }
  
  if (threadsPerBlock % gpuProps.warpSize)
    threadsPerBlock = gpuProps.warpSize * (threadsPerBlock / gpuProps.warpSize + 1);

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  printf("Blocks %u Threads %u Multiprocs: %u\n", numberOfBlocks, threadsPerBlock, gpuProps.multiProcessorCount);
  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
