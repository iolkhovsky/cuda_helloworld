#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void initWith(float num, float *a, int N)
{
  int stride = blockDim.x * gridDim.x;
  for(int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += stride)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

hipDeviceProp_t getProps() {
  int deviceId;
  hipGetDevice(&deviceId);
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);
  return props;
}

int main()
{
  const int N = 2<<24;
  size_t size = N * sizeof(float);
  
  int deviceId;
  hipGetDevice(&deviceId);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);
  
  hipMemPrefetchAsync(a, size, deviceId);
  hipMemPrefetchAsync(b, size, deviceId);
  hipMemPrefetchAsync(c, size, deviceId);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 1;
  numberOfBlocks = 1;
  hipDeviceProp_t gpuProps = getProps();
  int threads_per_sm = N / gpuProps.multiProcessorCount;
  if (N % gpuProps.multiProcessorCount)
      threads_per_sm++;
      
  if (threads_per_sm > 1024) {
      int k = threads_per_sm / 1024;
      if (threads_per_sm % 1024)
          k++;
      numberOfBlocks = k * gpuProps.multiProcessorCount;
      threadsPerBlock = N / numberOfBlocks;
      if (N % numberOfBlocks)
          threadsPerBlock++;
  } else {
      threadsPerBlock = threads_per_sm;
      numberOfBlocks = gpuProps.multiProcessorCount;
  }
  
  if (threadsPerBlock % gpuProps.warpSize)
    threadsPerBlock = gpuProps.warpSize * (threadsPerBlock / gpuProps.warpSize + 1);

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  printf("Blocks %u Threads %u Multiprocs: %u\n", numberOfBlocks, threadsPerBlock, gpuProps.multiProcessorCount);
  
  initWith<<<numberOfBlocks, threadsPerBlock>>>(3, a, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(4, b, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(0, c, N);
  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  hipMemPrefetchAsync(c, size, hipCpuDeviceId);
  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
